#include "hip/hip_runtime.h"
%%cu
#include<stdio.h>
#include<stdlib.h>
#include<string.h>
#define N 3
#define blocksize 3

float U[N*N],R[N*N],M[N*N],E[N*N],B[N*N],C[N*N],Final[N*N];
float Rinv[N*N],Uinv[N*N],REC[N*N];


__global__ void matCopy(float*a, float*b)        //Copies second matrix to first matrix
{
  int i = threadIdx.x;
    int j = blockIdx.x;
        b[i*N+j]=a[i*N+j];
}

__global__ void roundOff(float *mat)
{
  int i = threadIdx.x;
    int j = blockIdx.x;
            if(mat[i*N+j]>=0)
            mat[i*N+j]=(int)(mat[i*N+j]+0.5);
            else
            mat[i*N+j]=(int)(mat[i*N+j]-0.5);    
}
__global__ void matMul(float *a, float *b,float *c,int n){
    int i = threadIdx.x;
    int j = blockIdx.x;
    c[i*N+j]=0;
        for(int k=0;k<n;k++)
            c[i*N+j] += a[i*N+k]*b[k*N+j];
}


__global__ void PrintInverse(float *ar)
{
  int i = threadIdx.x;
    int j = blockIdx.x;
            printf("%f ", ar[i*N+j]);
    
}
__global__ void add(float *a, float *b){
    int i = threadIdx.x;
    int j = blockIdx.x;
        a[i*N+j] += b[i*N+j];
}


__global__ void nodiag_normalize(float *A, float *I, int n, int i){            
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < n && y < n)
if (x == i && x!=y){
I[x*n + y] /= A[i*n + i];
A[x*n + y] /= A[i*n + i];
}

}

__global__ void diag_normalize( float *A, float *I, int n, int i){            
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < n && y < n)
if (x == y && x == i){
I[x*n + y] /= A[i*n + i];
A[x*n + y] /= A[i*n + i];
}

}

__global__ void gaussjordan( float*A,  float*I, int n, int i)
{
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < n && y < n){
if (x != i){
I[x*n + y] -= I[i*n + y] * A[x*n + i];
if (y != i){
A[x*n + y] -= A[i*n + y] * A[x*n + i];
}    
}
}

}

__global__ void set_zero( float*A, float*I, int n, int i){
int x = blockIdx.x * blockDim.x + threadIdx.x;
int y = blockIdx.y * blockDim.y + threadIdx.y;

if (x < n && y < n){
if (x != i){
if (y == i){
A[x*n + y] = 0;
}
}
}
}


int main(void)
{
    
    for(int i=0;i<N;i++)
        for(int j=0; j<N; j++)
            {
          if(i==j)
                     U[i*N+j]=1.0;
            else
             U[i*N+j]=0.0;
      }

float time;
hipError_t err;
hipEvent_t start, stop;
hipEventCreate(&start);
hipEventCreate(&stop);
    
  for(int i=0;i<N;i++)
        for(int j=0; j<N; j++)
                {
                    R[i*N+j]=rand()%256;
                    M[i*N+j]=rand()%256;
                    E[i*N+j]=rand()%2;
                }    
    float *d_r, *d_m, *d_e,*d_b,*d_c,*d_rinv,*d_I,*d_uinv,*d_rec;
    float*d_final;
  hipMalloc((void**)&d_r, N*N*sizeof(float));
    hipMalloc((void**)&d_m, N*N*sizeof(float));
    hipMalloc((void**)&d_e, N*N*sizeof(float));
    hipMalloc((void**)&d_b, N*N*sizeof(float));
    hipMalloc((void**)&d_c, N*N*sizeof(float));
    hipMalloc((void**)&d_I, N*N*sizeof(float));
    hipMalloc((void**)&d_rinv, N*N*sizeof(float));
    hipMalloc((void**)&d_uinv, N*N*sizeof(float));
    hipMalloc((void**)&d_rec, N*N*sizeof(float));
    hipMalloc((void**)&d_final, N*N*sizeof(float));

hipMemcpy(d_I, U, N*N*sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(d_r, R, N*N*sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(d_b, B, N*N*sizeof(float) , hipMemcpyHostToDevice);

  hipEventRecord(start, 0);
     matMul<<<N,N>>>(d_I,d_r, d_b, N);
hipMemcpy(B,d_b,  N*N*sizeof(float) , hipMemcpyDeviceToHost);
printf("\nB:");
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
            printf("%f ",B[i*N+j]);
        printf("\n");
    }

    printf("\n\n");
hipMemcpy(d_m, M, N*N*sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(d_c, C, N*N*sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(d_b, B, N*N*sizeof(float) , hipMemcpyHostToDevice);

    
     matMul<<<N,N>>>(d_m,d_b, d_c, N);
  hipMemcpy(C,d_c,  N*N*sizeof(float) , hipMemcpyDeviceToHost);
printf("\nC:");
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
            printf("%f ",C[i*N+j]);
        printf("\n");
    }
 hipMemcpy(d_c, C, N*N*sizeof(float) , hipMemcpyHostToDevice);
 hipMemcpy(d_e, E, N*N*sizeof(float) , hipMemcpyHostToDevice);
   
     add<<<N,N>>>(d_c,d_e);
    
   hipMemcpy(C,d_c,  N*N*sizeof(float) , hipMemcpyDeviceToHost);

//END OF ENCRYPTION   
 printf("Rinv:\n");
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
         { Rinv[i*N+j]=R[i*N+j];


            
printf("%f ",Rinv[i*N+j]);
   }    
 printf("\n");
    }   

//hipMemcpy(d_r, R, N*N*sizeof(float) , hipMemcpyHostToDevice);
//hipMemcpy(d_rinv, Rinv, N*N*sizeof(float) , hipMemcpyHostToDevice);
//matCopy<<<N,N>>>(d_r,d_rinv);
//hipMemcpy(Rinv,d_rinv, N*N*sizeof(float) , hipMemcpyDeviceToHost);



 hipMemcpy(d_rinv, Rinv, N*N*sizeof(float) , hipMemcpyHostToDevice);


 hipMemcpy(d_I, U, N*N*sizeof(float) , hipMemcpyHostToDevice);
dim3 threadsPerBlock(blocksize, blocksize);
dim3 numBlocks((N + blocksize - 1) / blocksize, (N + blocksize - 1) / blocksize);

for (int i = 0; i<N; i++){
nodiag_normalize << <numBlocks, threadsPerBlock >> >(d_rinv, d_I, N, i);
diag_normalize << <numBlocks, threadsPerBlock >> >(d_rinv, d_I, N, i);
gaussjordan << <numBlocks, threadsPerBlock >> >(d_rinv, d_I, N, i);
set_zero << <numBlocks, threadsPerBlock >> >(d_rinv, d_I, N, i);
}
    hipMemcpy(Rinv,d_I, N*N*sizeof(float) , hipMemcpyDeviceToHost);
printf("***********After inverse:Rinv:*****************");
for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
            printf("%f ",Rinv[i*N+j]);
        printf("\n");
    }
 //hipMemcpy(d_I, U, N*N*sizeof(float) , hipMemcpyHostToDevice);
 hipMemcpy(d_uinv,Uinv, N*N*sizeof(float) , hipMemcpyHostToDevice);
 matCopy<<<N,N>>>(d_I,d_uinv);
   hipMemcpy(Uinv,d_uinv, N*N*sizeof(float) , hipMemcpyDeviceToHost);
hipMemcpy(d_uinv,Uinv, N*N*sizeof(float) , hipMemcpyHostToDevice);
for (int i = 0; i<N; i++){
nodiag_normalize << <numBlocks, threadsPerBlock >> >(d_uinv, d_I, N, i);
diag_normalize << <numBlocks, threadsPerBlock >> >(d_uinv, d_I, N, i);
gaussjordan << <numBlocks, threadsPerBlock >> >(d_uinv, d_I, N, i);
set_zero << <numBlocks, threadsPerBlock >> >(d_uinv, d_I, N, i);
}
    hipMemcpy(Uinv,d_uinv, N*N*sizeof(float) , hipMemcpyDeviceToHost);
    
hipMemcpy(d_rinv, Rinv, N*N*sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(d_c, C, N*N*sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(d_rec, REC, N*N*sizeof(float) , hipMemcpyHostToDevice);

    
     matMul<<<N,N>>>(d_c,d_rinv, d_rec, N);
  hipMemcpy(REC,d_rec,  N*N*sizeof(float) , hipMemcpyDeviceToHost);
printf("\nREC:");
    
        for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
            printf("%f ",REC[i*N+j]);
        printf("\n");
    }

    printf("\n\n");
  hipMemcpy(d_rec, REC, N*N*sizeof(float) , hipMemcpyHostToDevice);
  roundOff<<<N,N>>>(d_rec);
    
    hipMemcpy(REC,d_rec,  N*N*sizeof(float) , hipMemcpyDeviceToHost);
 printf("\nAfter Roundoff REC:");
    
        for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
            printf("%f ",REC[i*N+j]);
        printf("\n");
    }

    printf("\n\n");
  hipMemcpy(d_uinv, Uinv, N*N*sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(d_final, Final, N*N*sizeof(float) , hipMemcpyHostToDevice);
hipMemcpy(d_rec, REC, N*N*sizeof(float) , hipMemcpyHostToDevice);

    
     matMul<<<N,N>>>(d_rec,d_uinv, d_final, N);
  hipMemcpy(Final,d_final,  N*N*sizeof(float) , hipMemcpyDeviceToHost);
printf("\n Final:");
    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
            printf("%f ",Final[i*N+j]);
        printf("\n");
    }
    printf("\n\n");


    for(int i=0;i<N;i++)
    {
        for(int j=0;j<N;j++)
            printf("%f ",Final[i*N+j]-M[i*N+j]);
        printf("\n");
    }

    printf("\n\n");  
   
hipEventRecord(stop, 0);
hipEventSynchronize(stop);
hipEventElapsedTime(&time, start, stop);
hipEventDestroy(start);
hipEventDestroy(stop);
   printf("\nTime taken is: %f",time);
   
hipFree(d_I);
hipFree(d_m);
hipFree(d_r);
hipFree(d_b);
hipFree(d_rec);
hipFree(d_rinv);
hipFree(d_final);
hipFree(d_uinv);
hipFree(d_e);
return 0;
}

/*
'\nB:103.000000 115.000000 74.000000 \n205.000000 242.000000 70.000000 \n84.000000 232.000000 118.000000 \n\n\n\nC:56823.000000 97124.000000 48170.000000 \n81029.000000 110900.000000 45966.000000 \n80459.000000 105302.000000 45142.000000 \nRinv:\n103.000000 115.000000 74.000000 \n205.000000 242.000000 70.000000 \n84.000000 232.000000 118.000000 \n***********After inverse:Rinv:*****************0.010454 0.003054 -0.008368 \n-0.015542 0.005040 0.006757 \n0.023116 -0.012084 0.001147 \n\nREC:198.017899 80.995987 235.999512 \n185.994766 251.008133 123.998329 \n247.994797 231.008118 89.998367 \n\n\n\nAfter Roundoff REC:198.000000 81.000000 236.000000 \n186.000000 251.000000 124.000000 \n248.000000 231.000000 90.000000 \n\n\n\n Final:198.000000 81.000000 236.000000 \n186.000000 251.000000 124.000000 \n248.000000 231.000000 90.000000 \n\n\n\nDifference0.000000 0.000000 0.000000 \n0.000000 0.000000 0.000000 \n0.000000 0.000000 0.000000 \n\n\n\nTime taken is: 0.688544'
*/
